#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>


#include <iostream>
#include <cmath>
#include <vector>



#define mu 0.05f
#define sigma .2f
#define timespan 252.0f


#define TRIALS 10000
#define numThreads 512


#include <random>

__global__ void europeanOption(
	int size, int iterations,
	float *d_price, float initialPrice, float strikePrice,
	hiprandState_t *d_state)
{
	int tid = threadIdx.x + blockDim.x * blockIdx.x;
	/*std::vector<double> correlated;
	double rho = .6;*/

	float spot_normals;
	int temp = 0;
	
	std::random_device rd;
	std::mt19937 e2(rd());
	std::normal_distribution<> dist(0, 1);

	if (tid < size)
	{

		for (int i = 0; i < iterations; i++)
		{
			initialPrice *= 1 + mu / timespan + hiprand_normal(&d_state[tid])*sigma / sqrt(timespan); // initial code
			/*for (int n = 0; n < 100; ++n) {
				temp = std::round(dist(e2));
			}*/
			/*
			1) Use the correlated brownian motion to create the vol_path
			2) Use that vol path to create the spot_path
			3) Use the vol_path and spot_path to create asset path
			4)
			*/
				

			//correlated[i] = rho * (hiprand_normal(&d_state[tid]))[i] + correlated[i] * sqrt(1 - rho*rho);

			/*for (int i = 0; i<vals; i++) {
				correlated[i] = rho * (spot_normals)[i] + correlated[i] * sqrt(1 - rho*rho);
			}*/
		}

		d_price[tid] = initialPrice - strikePrice;
		if (d_price[tid] < 0)
		{
			d_price[tid] = 0;
		}
	}

}

__global__ void init(
	unsigned int seed,
	hiprandState_t *d_state)
{
	hiprand_init(
		seed,
		threadIdx.x + blockDim.x * blockIdx.x,
		0,
		&d_state[threadIdx.x + blockDim.x * blockIdx.x]);
}


int main()
{

	float *h_prices, *d_prices;

	h_prices = new float[TRIALS];
	hipMalloc((void**)&d_prices, TRIALS*sizeof(float));

	hiprandState_t *d_state;
	hipMalloc((void**)&d_state, TRIALS * sizeof(hiprandState_t));

	init << < (TRIALS - numThreads - 1) / numThreads, numThreads >> >(time(0), d_state);

	europeanOption << <(TRIALS - numThreads - 1) / numThreads, numThreads >> >(
		TRIALS, 252,
		d_prices, 100.0f, 100.0f,
		d_state);

	hipMemcpy(h_prices, d_prices, TRIALS*sizeof(float), hipMemcpyDeviceToHost);

	float price = 0;

	int count = 0;

	for (int i = 0; i < TRIALS; i++)
	{
		price += h_prices[i];
		if (h_prices[i] != 0)
		{
			count += 1;
		}
	}

	price /= TRIALS;

	std::cout << "The Theoretical Price of the Option is " << price << "." << std::endl;

	std::cout << "Count is " << count << "." << std::endl;

	delete[] h_prices;
	hipFree(d_state); hipFree(d_prices);

	hipDeviceReset();

	int i;
	std::cin >> i;
	return 0;
}