#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>


#include <iostream>
#include <cmath>
#include <vector>



#define mu 0.05f
#define sigma .2f
#define timespan 252.0f


#define TRIALS 10000
#define numThreads 512


__global__ void europeanOption(
	int size, int iterations,
	float *d_price, float initialPrice, float strikePrice,
	hiprandState_t *d_state)
{
	int tid = threadIdx.x + blockDim.x * blockIdx.x;
	std::vector<double> correlated;
	double rho = .6;
	if (tid < size)
	{

		for (int i = 0; i < iterations; i++)
		{
			initialPrice *= 1 + mu / timespan + hiprand_normal(&d_state[tid])*sigma / sqrt(timespan);

			correlated[i] = rho * (hiprand_normal(&d_state[tid]))[i] + correlated[i] * sqrt(1 - rho*rho);

			/*for (int i = 0; i<vals; i++) {
				correlated[i] = rho * (spot_normals)[i] + correlated[i] * sqrt(1 - rho*rho);
			}*/
		}

		d_price[tid] = initialPrice - strikePrice;
		if (d_price[tid] < 0)
		{
			d_price[tid] = 0;
		}
	}

}

__global__ void init(
	unsigned int seed,
	hiprandState_t *d_state)
{
	hiprand_init(
		seed,
		threadIdx.x + blockDim.x * blockIdx.x,
		0,
		&d_state[threadIdx.x + blockDim.x * blockIdx.x]);
}


int main()
{

	float *h_prices, *d_prices;

	h_prices = new float[TRIALS];
	hipMalloc((void**)&d_prices, TRIALS*sizeof(float));

	hiprandState_t *d_state;
	hipMalloc((void**)&d_state, TRIALS * sizeof(hiprandState_t));

	init << < (TRIALS - numThreads - 1) / numThreads, numThreads >> >(time(0), d_state);

	europeanOption << <(TRIALS - numThreads - 1) / numThreads, numThreads >> >(
		TRIALS, 252,
		d_prices, 100.0f, 100.0f,
		d_state);

	hipMemcpy(h_prices, d_prices, TRIALS*sizeof(float), hipMemcpyDeviceToHost);

	float price = 0;

	int count = 0;

	for (int i = 0; i < TRIALS; i++)
	{
		price += h_prices[i];
		if (h_prices[i] != 0)
		{
			count += 1;
		}
	}

	price /= TRIALS;

	std::cout << "The Theoretical Price of the Option is " << price << "." << std::endl;

	std::cout << "Count is " << count << "." << std::endl;

	delete[] h_prices;
	hipFree(d_state); hipFree(d_prices);

	hipDeviceReset();

	int i;
	std::cin << i;
	return 0;
}